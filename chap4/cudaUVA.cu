#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void sum(int *a, int *b, int *c, int n) {
    int tid = threadIdx.x;
    if(tid < n) c[tid] = a[tid] + b[tid];
}

int main() {

    int n = 10;
    int *a = NULL, *b = NULL, *c = NULL;

    hipHostAlloc(&a, n * sizeof(int), hipHostMallocMapped);
    hipHostAlloc(&b, n * sizeof(int), hipHostMallocMapped);
    hipHostAlloc(&c, n * sizeof(int), hipHostMallocMapped);

    for(int i = 0 ; i < n; i++) {
        a[i] = b[i] = i;
    }

    sum<<<1, n>>>(a, b, c, n);
    hipDeviceSynchronize();
    for(int i = 0; i < n; i++) {
        printf("%d ", c[i]);
    }

    hipFree(a);
    hipFree(b);
    hipFree(c);

    hipDeviceReset();
    return 0;
}