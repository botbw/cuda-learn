#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ float devA;

__global__ void addDevA() {
    devA += 10;
}

int main() {
    float a = 2.333;
    printf("Value %f\n", a);
    hipMemcpyToSymbol(HIP_SYMBOL(devA), &a, sizeof(a));
    addDevA<<<1, 1>>>();
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&a, HIP_SYMBOL(devA), sizeof(a));
    printf("Value %f\n", a);

    hipDeviceReset();
    return 0;
}