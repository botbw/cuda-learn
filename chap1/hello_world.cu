
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_world() {
    printf("Hello from GPU\n");
}

int main() {
    printf("Hello from CPU");
    hello_world<<<1,10>>>();
    hipDeviceReset();
    return 0;
}
